#include "hip/hip_runtime.h"
#include <thread>
#include "platform.hpp"
#include "../algorithms/algorithm.hpp"
#include "../constants.hpp"
#include ALGORITHM_IMPL

__global__
void resize(u8* input, u8* output) {
  uint x = (blockIdx.x * blockDim.x) + threadIdx.x;
  uint y = (blockIdx.y * blockDim.y) + threadIdx.y;
  write_pixel(x, y, input, output);
}

u64 platform_run(hemi::Array<u8> &input, hemi::Array<u8> &output) {
  u8* input_ptr = input.devicePtr();
  u8* output_ptr = output.devicePtr();

  dim3 threadsPerBlock(16, 16);
  dim3 numBlocks((OUT_WIDTH/threadsPerBlock.x) + 1, (OUT_HEIGHT/threadsPerBlock.y) + 1);

  const auto before = std::chrono::system_clock::now();
  resize<<<numBlocks, threadsPerBlock>>>(input_ptr, output_ptr);
  hipDeviceSynchronize();
  const auto after = std::chrono::system_clock::now();

  return std::chrono::duration_cast<std::chrono::microseconds>(after - before).count();
}
